#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define sign(x) (x>0)-(x<0)

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char *msg, int line = -1) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        if (line >= 0) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


/*****************
 * Configuration *
 *****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

/*! Mass of one "planet." */
#define planetMass 3e8f

/*! Mass of the "star" at the center. */
#define starMass 5e10f

/*! Size of the starting area in simulation space. */
const float scene_scale = 1e2;


/***********************************************
 * Kernel state (pointers are device pointers) *
 ***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

glm::vec3 *dev_pos;
glm::vec3 *dev_vel;
glm::vec3 *dev_acc;


/******************
 * initSimulation *
 ******************/

__host__ __device__ unsigned int hash(unsigned int a) {
    a = (a + 0x7ed55d16) + (a << 12);
    a = (a ^ 0xc761c23c) ^ (a >> 19);
    a = (a + 0x165667b1) + (a << 5);
    a = (a + 0xd3a2646c) ^ (a << 9);
    a = (a + 0xfd7046c5) + (a << 3);
    a = (a ^ 0xb55a4f09) ^ (a >> 16);
    return a;
}

/**
 * Function for generating a random vec3.
 */
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
    thrust::default_random_engine rng(hash((int)(index * time)));
    thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

    return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
 * CUDA kernel for generating planets with a specified mass randomly around the star.
 */
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale, float mass) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        glm::vec3 rand = generateRandomVec3(time, index);
        arr[index].x = scale * rand.x;
        arr[index].y = scale * rand.y;
        arr[index].z = 0.1 * scale * sqrt(rand.x * rand.x + rand.y * rand.y) * rand.z;
    }
}

/**
 * CUDA kernel for generating velocities in a vortex around the origin.
 * This is just to make for an interesting-looking scene.
 */
__global__ void kernGenerateCircularVelArray(int time, int N, glm::vec3 * arr, glm::vec3 * pos) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        glm::vec3 R = glm::vec3(pos[index].x, pos[index].y, pos[index].z);
        float r = glm::length(R) + EPSILON;
        float s = sqrt(G * starMass / r);
        glm::vec3 D = glm::normalize(glm::cross(R / r, glm::vec3(0, 0, 1)));
        arr[index].x = s * D.x;
        arr[index].y = s * D.y;
        arr[index].z = s * D.z;
    }
}

/**
 * Initialize memory, update some globals
 */
void Nbody::initSimulation(int N) {
	//N = 5;
    numObjects = N;
    dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

    hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
    checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

    hipMalloc((void**)&dev_vel, N * sizeof(glm::vec3));
    checkCUDAErrorWithLine("hipMalloc dev_vel failed!");

    hipMalloc((void**)&dev_acc, N * sizeof(glm::vec3));
    checkCUDAErrorWithLine("hipMalloc dev_acc failed!");

    kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale, planetMass);
    checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

    kernGenerateCircularVelArray<<<fullBlocksPerGrid, blockSize>>>(2, numObjects, dev_vel, dev_pos);
    checkCUDAErrorWithLine("kernGenerateCircularVelArray failed!");

    hipDeviceSynchronize();
}


/******************
 * copyPlanetsToVBO *
 ******************/

/**
 * Copy the planet positions into the VBO so that they can be drawn by OpenGL.
 */
__global__ void kernCopyPlanetsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale = -1.0f / s_scale;

    if (index < N) {
        vbo[4 * index + 0] = pos[index].x * c_scale;
        vbo[4 * index + 1] = pos[index].y * c_scale;
        vbo[4 * index + 2] = pos[index].z * c_scale;
        vbo[4 * index + 3] = 1;
    }
}

/**
 * Wrapper for call to the kernCopyPlanetsToVBO CUDA kernel.
 */
void Nbody::copyPlanetsToVBO(float *vbodptr) {
    dim3 fullBlocksPerGrid((int)ceil(float(numObjects) / float(blockSize)));

    kernCopyPlanetsToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr, scene_scale);
    checkCUDAErrorWithLine("copyPlanetsToVBO failed!");

    hipDeviceSynchronize();
}


/******************
 * stepSimulation *
 ******************/

 __device__ glm::vec3 single_accelerate(glm::vec3 this_planet, glm::vec3 other_planet, int isPlanet){
	// isPlanet: 1 if true, else 0

	glm::vec3 dir = glm::normalize(other_planet - this_planet);
	float r = glm::length(other_planet - this_planet);
	float r2 = r*r + EPSILON;
	float mass = planetMass*isPlanet + starMass*(1-isPlanet);
	float s = (G * mass / r2);
	
	glm::vec3 g;
	g.x = dir.x * s;
	g.y = dir.y * s;
	g.z = dir.z * s;
	return g;
 }

/**
 * Compute the acceleration on a body at `my_pos` due to the `N` bodies in the array `other_planets`.
 */
__device__  glm::vec3 accelerate(int N, int iSelf, glm::vec3 this_planet, const glm::vec3 *other_planets) {
    // TODO: Compute the acceleration on `my_pos` due to:
    //   * The star at the origin (with mass `starMass`)
    //   * All of the *other* planets (with mass `planetMass`)
    // Return the sum of all of these contributions.

    // HINT: You may want to write a helper function that will compute the acceleration at
    //   a single point due to a single other mass. Be careful that you protect against
    //   division by very small numbers.
    // HINT: Use Newtonian gravitational acceleration:
    //       G M
    //  g = -----
    //       r^2
    //  where:
    //    * G is the universal gravitational constant (already defined for you)
    //    * M is the mass of the other object
    //    * r is the distance between this object and the other object

	glm::vec3 single_acc;
	glm::vec3 total_acc(0.0f, 0.0f, 0.0f);

	total_acc += single_accelerate(this_planet, glm::vec3(0.0f), 0);
	for (int i=0; i<N; ++i){
		if(i == iSelf){
			continue;
		}
		single_acc = single_accelerate(this_planet, other_planets[i], 1);
		total_acc += single_acc;
	}
    
    return total_acc;
}

/**
 * For each of the `N` bodies, update its acceleration.
 * Compute the total instantaneous acceleration using `accelerate`, then store that into `acc`.
 */
__global__ void kernUpdateAcc(int N, float dt, const glm::vec3 *pos, glm::vec3 *acc) {
    // TODO: implement updateAccArray.
    // This function body runs once on each CUDA thread.
    // To avoid race conditions, each instance should only write ONE value to `acc`!

	int i = threadIdx.x + (blockIdx.x * blockDim.x);

	//int i = threadIdx.x;
	if (i < N){
		acc[i] = accelerate(N,i,pos[i],pos);
	}

}

/**
 * For each of the `N` bodies, update its velocity, then update its position, using a
 * simple Euler integration scheme. Acceleration must be updated before calling this kernel.
 */
__global__ void kernUpdateVelPos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel, const glm::vec3 *acc) {
    // TODO: implement updateVelocityPosition
	//int i = threadIdx.x;
	int i = threadIdx.x + (blockIdx.x * blockDim.x);

	if (i < N){
		vel[i] = vel[i] + acc[i]*dt;
		pos[i] = pos[i] + vel[i]*dt;
	}

}

/**
 * Step the entire N-body simulation by `dt` seconds.
 */
void Nbody::stepSimulation(float dt) {
    // TODO: Using the CUDA kernels you wrote above, write a function that
    // calls the kernels to perform a full simulation step.

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	// Kernel acc update
	kernUpdateAcc<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_acc);

	// Kernel vel update
	kernUpdateVelPos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel, dev_acc);
}
